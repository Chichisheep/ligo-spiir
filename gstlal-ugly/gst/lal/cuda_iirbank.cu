extern "C"
{
#include "cuda_iirbank.h"
}
int cudaUtils()
{
  printf("nothing\n");
  char *data;
  char *cudaData;
  data = (char*) malloc (12 * sizeof(char));
  strncpy(data, "hello world!", 12);
  data[11] = '\0'; 
  data[0] = 'b';

  hipMalloc( (void **)&cudaData, 12 * sizeof(char) );
  hipMemcpy( cudaData, data, 12 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy( data, cudaData, 12 * sizeof(char), hipMemcpyDeviceToHost);
  printf("%s\n", data);
	
	return 0;
}
